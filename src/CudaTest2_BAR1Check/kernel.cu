﻿
#include "hip/hip_runtime.h"
#include "nvml.h"


//#include "nv-p2p.h"

#include <stdio.h>

int main()
{
    hipError_t status = hipSetDeviceFlags(hipDeviceMapHost);
    status = hipSetDevice(0);
    CUDA_POINTER_ATTRIBUTE_P2P_TOKENS tokens;
    hipDeviceptr_t ptr = 0;
    hipError_t result = hipSuccess;
    //cuDevicePrimaryCtxRetain();
    result = hipMalloc(&ptr, 4096);
    if(result == hipSuccess)
    {
        result = hipPointerGetAttribute(&tokens, HIP_POINTER_ATTRIBUTE_P2P_TOKENS, ptr);
        hipFree(ptr);
    }

    hipDeviceSynchronize();
    hipDeviceReset();
    return 0;
}
