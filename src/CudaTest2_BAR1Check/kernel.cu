﻿
#include "hip/hip_runtime.h"
#include "nvml.h"
#include "hip/hip_runtime.h"
#include ""
//#include "nv-p2p.h"

#include <stdio.h>

int main()
{
    //hipError_t status = hipSetDeviceFlags(hipDeviceMapHost);
    hipError_t status = hipSetDevice(0);
    hipDevice_t device;

    CUDA_POINTER_ATTRIBUTE_P2P_TOKENS tokens;
    hipDeviceptr_t ptr = 0;
    hipError_t result = hipSuccess;
    // Get handle for device 0
    result = hipDeviceGet(&device, 0);
    // Create context
    hipCtx_t context;
    hipCtxCreate(&context, 0, device);

    //hipDevicePrimaryCtxRetain();
    result = hipMalloc(&ptr, 4096);
    hipDeviceSynchronize();
    if(result == hipSuccess)
    {
        result = hipPointerGetAttribute(&tokens, HIP_POINTER_ATTRIBUTE_P2P_TOKENS, ptr);
        if(result == hipSuccess)
            printf("p2pToken=%lld, vaSpaceToken=%d\n", tokens.p2pToken, tokens.vaSpaceToken);
        else
            printf("hipPointerGetAttribute failed (%d)\n", result);

        hipDeviceSynchronize();
        hipFree(ptr);
    }
    else
        printf("hipMalloc failed (%d)\n", result);

    hipDeviceSynchronize();
    hipCtxDestroy(context);
    hipDeviceReset();
    return 0;
}
